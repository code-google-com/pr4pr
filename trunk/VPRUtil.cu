#include "hip/hip_runtime.h"
﻿#include "VPRUtil.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>

#include <boost/progress.hpp>
#include <boost/scoped_array.hpp>


struct TentKernel
{
	TentKernel(const float3& TheSP)
	{
		SP.x = TheSP.x;
		SP.y = TheSP.y;
		SP.z = TheSP.z;
	}
	
	__host__ __device__
	float operator()(const float3 P, const float Sum)
	{
		return Sum + (1.0f-abs(P.x-SP.x))*(1.0f-abs(P.y-SP.y))*(1.0-abs(P.z-SP.z));
	}
	
	float3 SP;
};

bool MakeTent(const size_t Len, const float3* Points, const int Res[3], const float3& Min, const float3& Max, float* Data)
{
	try
	{
		float3 Extent;
		Extent.x = Max.x - Min.x;
		Extent.y = Max.y - Min.y;
		Extent.z = Max.z - Min.z;
		
		float3 Delta;
		Delta.x = Extent.x / (float)Res[0];
		Delta.y = Extent.y / (float)Res[1];
		Delta.z = Extent.z / (float)Res[2];
		
		std::cout<<"MakeTent "<<Res[0]<<'\t'<<Res[1]<<'\t'<<Res[2]<<std::endl;
		const int TotalCount = Res[0]*Res[1]*Res[2];
		
		boost::scoped_array<float3> GridPosHst(new float3[TotalCount]);
		thrust::device_vector<float3> GridPosDev(TotalCount);
		thrust::device_vector<float> TentFuncDev(TotalCount,0.0f);
		
		for( int z=0; z<Res[2]; ++z )
		{
			int ZOffset = z*Res[0]*Res[1];
			for( int y=0; y<Res[1]; ++y )
			{
				int YOffset = y*Res[0];
				for( int x=0; x<Res[0]; ++x )
				{
					int XOffset = x;
					float3 P;
					P.x = ( (float)x+0.5f )*Delta.x + Min.x;
					P.y = ( (float)y+0.5f )*Delta.y + Min.y;
					P.z = ( (float)z+0.5f )*Delta.z + Min.z;
					GridPosHst[ZOffset+YOffset+XOffset] = P;
				}
			}
		}
		thrust::copy(GridPosHst.get(),GridPosHst.get()+TotalCount,GridPosDev.begin());
		GridPosHst.reset();

		boost::progress_display Progress(Len);
		boost::progress_timer Timer;
		
		std::cout<<"ParticleResolver : MakeTent Start to Calculate the Tent Function"<<std::endl;
		for( size_t i=0; i<Len; ++i )
		{
			thrust::transform(GridPosDev.begin(),GridPosDev.end(),TentFuncDev.begin(),TentFuncDev.end(),TentKernel(Points[i]));
			++Progress;
		}
		thrust::copy(TentFuncDev.begin(),TentFuncDev.end(),Data);

	}catch(...)
	{
		return false;
	}
	
	
	return true;
}
