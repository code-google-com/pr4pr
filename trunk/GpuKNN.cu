#include "hip/hip_runtime.h"
﻿#include "PR.h"

#include "GpuKNN.h"

#include <thrust/transform.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sort.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <iostream>

NAMESPACE_PR_BEGIN

struct SqrtDist
{
	float3 Center;
	
	SqrtDist(const float3& P)
	{
		Center.x = P.x;
		Center.y = P.y;
		Center.z = P.z;
	}
    __host__ __device__ float operator()(const float3& P) const
    {
        float a = P.x - Center.x;
        float b = P.y - Center.y;
        float c = P.z - Center.z;
        return a*a + b*b + c*c;
    }
};

GpuKNN::GpuKNN(const size_t Count, const Float3* Points)
{
	mPoints.resize(Count);
	mIndices.resize(Count);
	const float3* p = (const float3*)(Points);
	thrust::copy(p,p+Count,mPoints.begin());
	thrust::sequence(mIndices.begin(), mIndices.end());
}

void GpuKNN::Lookup(const Float3& P, const float SqrtR, const size_t Count, int* NNIndices)
{
	thrust::device_vector<float> Dist(mPoints.size());
	float3 P2;
	P2.x = P.x;
	P2.y = P.y;
	P2.z = P.z;
	thrust::transform(mPoints.begin(), mPoints.end(),  Dist.begin(), SqrtDist(P2));
	thrust::sort_by_key(Dist.begin(),Dist.end(), mIndices.begin());
	thrust::copy(mIndices.begin(), mIndices.begin()+Count, NNIndices);
}

NAMESPACE_PR_END
