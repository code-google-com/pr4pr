#include "hip/hip_runtime.h"
﻿#include "GpuKNN.h"

#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/transform.h>
#include <iostream>

struct SqrtDist
{
	float3 Center;
	
	SqrtDist(const Float3& P)
	{
		Center.x = P.x;
		Center.y = P.y;
		Center.z = P.z;
	}
    __host__ __device__ float operator()(const float3& P) const
    {
        float a = P.x - Center.x;
        float b = P.y - Center.y;
        float c = P.z - Center.z;
        return a*a + b*b + c*c;
    }
};

GpuKNN::GpuKNN(const size_t Count, const Float3* Points)
{
	mPoints.resize(Count);
	mPointIndices.resize(Count);
	float3* p = static_cast<float3*>(Points);
	thrust::copy(Points,Points+Count,mPoints.begin();
	thrust::sequence(mIndices.begin(), mIndices.end());
}

void GpuKNN::Lookup(const Float3& P, const float SqrtR, const size_t Count, int* NNIndices)
{
	thrust::device_vector<float> Dist(mPoints.size());
	thrust::transform(mPoints.begin(), mPoints.end(),  Dist.begin(), SqrtDist(P));
	thrust::sort_by_key(Dist.begin(),Dist.end(), mIndices.begin());
	thrust::copy(mIndices.begin(), mIndices.begin()+Count, NNIndices);
}
