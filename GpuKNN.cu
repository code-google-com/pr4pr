#include "hip/hip_runtime.h"
﻿/*
 * Copyright (c) 2010, Bo Zhou <Bo.Schwarzstein@gmail.com>
 * All rights reserved.
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the Bo Zhou, nor the names of its contributors
 *       may be used to endorse or promote products derived from this
 *       software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE REGENTS AND CONTRIBUTORS BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "PR.h"

#include "GpuKNN.h"

#include <thrust/transform.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sort.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <iostream>

NAMESPACE_PR_BEGIN

struct SqrtDist
{
	float3 Center;
	
	SqrtDist(const float3& P)
	{
		Center.x = P.x;
		Center.y = P.y;
		Center.z = P.z;
	}
    __host__ __device__ float operator()(const float3& P) const
    {
        float a = P.x - Center.x;
        float b = P.y - Center.y;
        float c = P.z - Center.z;
        return a*a + b*b + c*c;
    }
};

GpuKNN::GpuKNN(const size_t Count, const Float3* Points)
{
	mPoints.resize(Count);
	mIndices.resize(Count);
	const float3* p = (const float3*)(Points);
	thrust::copy(p,p+Count,mPoints.begin());
	thrust::sequence(mIndices.begin(), mIndices.end());
}

void GpuKNN::Lookup(const Float3& P, const float SqrtR, const size_t Count, int* NNIndices)
{
	thrust::device_vector<float> Dist(mPoints.size());
	float3 P2;
	P2.x = P.x;
	P2.y = P.y;
	P2.z = P.z;
	thrust::transform(mPoints.begin(), mPoints.end(),  Dist.begin(), SqrtDist(P2));
	thrust::sort_by_key(Dist.begin(),Dist.end(), mIndices.begin());
	thrust::copy(mIndices.begin(), mIndices.begin()+Count, NNIndices);
}

NAMESPACE_PR_END
